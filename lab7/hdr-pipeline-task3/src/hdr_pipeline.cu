#include "hip/hip_runtime.h"
#include <cstdint>
#include "utils/cuda/error.h"

#include "bloom_kernel.cuh"

unsigned int divup(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}


__device__ float tonemap(float value, float exposure) {
    float v = value * exposure;
    return (v * (0.9036f * v + 0.018f)) / (v * (0.8748f * v + 0.354f) + 0.14f);
}

__device__ float3 tonemap(float3 value, float exposure) {
    return { tonemap(value.x, exposure), tonemap(value.y, exposure), tonemap(value.z, exposure) };
}

__device__ float sRGB8(float color) {
    return color <= 0.0031308f ? color * 12.92f : 1.055f * pow(color, 1.0f / 2.4f) - 0.055f;
}

__device__ uint32_t norm(float color) {
    return static_cast<uint32_t>(min(max(color, 0.0f), 1.0f) * 255.0f);
}

__device__ uint32_t sRGB8(float3 color) {
    uint32_t r = norm(sRGB8(color.x));
    uint32_t g = norm(sRGB8(color.y));
    uint32_t b = norm(sRGB8(color.z));
    uint32_t a = 255;
    return (a << 24) | (b << 16) | (g << 8) | r;
}

__global__ void tonemap_kernel(uint32_t* out, const float* in, int width, int height, float exposure) {
    unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        float3 color = { in[4*(y*width + x) + 0], in[4*(y*width + x) + 1], in[4*(y*width + x) + 2] };

        uint32_t srgb = sRGB8(tonemap(color, exposure));

        out[y*width + x] = srgb; 
    }
}

__device__ float log_avg_lum;

__device__ float luminance(float3 color) {
    return 0.2126f * color.x + 0.7152f * color.y + 0.0722f * color.z;
}

__global__ void log_avg_lum_kernel(const float* in, int width, int height) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float3 color = { in[4 * (y * width + x) + 0], in[4 * (y * width + x) + 1], in[4 * (y * width + x) + 2] };

        float lum = luminance(color);

        atomicAdd(&log_avg_lum, log(lum));
    }
}

__device__ float calc_fade_contribution(float3 v, float exposure, float threshold){
    
    

    float res = powf(saturate((tonemap(v, exposure) - 0.8f * threshold) / 0.2f * threshold), 2);
    return res;
}

float saturate(float x){
    if(x < 0.0f){
        return 0.0f;
    } 
    if (x > 1.0f){
        return 1.0f;
    }
    else{
        return x;
    }
}


void bright_pass_x_y(float exposure, const float* in){
    const float threshold = 1.0f;
    // todo: cuda e
    calc_fade_contribution(in, exposure, threshold) * in;
}


void blurr(){

}

void tonemap(uint32_t* out, const float* in, int width, int height, float exposure, float brightpass_threshold) {
    int block_size_x = 32;
    int block_size_y = 8;

    dim3 block(block_size_x, block_size_y, 1);
    dim3 grid(divup(width, block.x), divup(height, block.y), 1);
    tonemap_kernel<<<grid, block>>>(out, in, width, height, exposure);
    throw_error(hipDeviceSynchronize());
    throw_error(hipPeekAtLastError());
}

float compute_avg_luminance(const float* in, int width, int height) {
    constexpr int block_size_x = 128;
    constexpr int block_size_y = 8;

    float luminance = 0;

    const dim3 block_size = { block_size_x , block_size_y };
    const dim3 num_blocks = { divup(width, block_size_x), divup(height, block_size_y) };

    hipMemcpyToSymbol(HIP_SYMBOL(log_avg_lum), &luminance, sizeof(float));

    log_avg_lum_kernel<<<num_blocks, block_size>>>(in, width, height);

    hipMemcpyFromSymbol(&luminance, HIP_SYMBOL(log_avg_lum), sizeof(float));

    return std::exp(luminance / (width * height));
}
